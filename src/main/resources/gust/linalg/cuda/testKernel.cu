
#include <hip/hip_runtime.h>
__device__ float f(float x);

extern "C"
__global__ void map_fun(float *A, int n)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx >= n) return;

    A[idx] = f(A[idx]);
}


__device__ float f(float x) { return x*x - 10.0; }